
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define N 1024
#define block_size 32

__global__ void MM(int *a, int *b, int *c){
    int row, col, my_x, my_y;
    row = threadIdx.y;
    col = threadIdx.x;
    my_y = blockIdx.y * blockDim.y + threadIdx.y;
    my_x = blockIdx.x * blockDim.x + threadIdx.x;

    int i, j;
    int result = 0;
    __shared__ int A_s[block_size][block_size];
    __shared__ int B_s[block_size][block_size];
    
    
    for (i = 0; i < N / block_size; i++) {
        A_s[row][col] = a[my_y * N + (i * blockDim.y + col)];
        B_s[row][col] = b[(i * blockDim.x + row) * N + my_x];
        __syncthreads();
        for (j = 0; j < block_size; j++) {
            result += A_s[row][j] * B_s[j][col];
        }
        __syncthreads();
    }
    c[my_y * N + my_x] = result;
    
    /*
    for (i = 0; i < N / block_size; i++) {
        A_s[row][col] = a[my_x * N + (i * blockDim.y + col)];
        B_s[row][col] = b[(i * blockDim.x + row) * N + my_y];
        __syncthreads();
        for (j = 0; j < block_size; j++) {
            result += A_s[row][j] * B_s[j][col];
        }
        __syncthreads();
    }
    c[my_x * N + my_y] = result;
    */
}

int main() {
    int i, j;
    int *a, *b, *c;
    a = (int *) malloc(sizeof(int) * N * N);
    b = (int *) malloc(sizeof(int) * N * N);
    c = (int *) malloc(sizeof(int) * N * N);
    
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            
            a[i * N + j] = 1;
            b[i * N + j] = 2;
            
            /*
            a[i * N + j] = j;
            b[i * N + j] = j + 1;
            */
        }
    }
    
    int *gpu_a, *gpu_b, *gpu_c;
    hipMalloc((void**)&gpu_a, sizeof(int) * N * N);
    hipMalloc((void**)&gpu_b, sizeof(int) * N * N);
    hipMalloc((void**)&gpu_c, sizeof(int) * N * N);
    
    hipMemcpy(gpu_a, a, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(int) * N * N, hipMemcpyHostToDevice);
    
    dim3 dimGrid(32, 32);
    dim3 dimBlock(32, 32);
    
    struct timespec start, stop;
    double time;
    
    if(clock_gettime(CLOCK_REALTIME, &start) == -1 ) {perror("clock gettime");}
    MM<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
    hipMemcpy(c, gpu_c, sizeof(int) * N * N, hipMemcpyDeviceToHost);
    if(clock_gettime(CLOCK_REALTIME, &stop) == -1 ) {perror("clock gettime");}
    
    time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec) / 1e9;
    printf("Execution time is %f ns\n", time * 1e9);
    printf("C[451][451] is %d\n", c[451 * N + 451]);
    
    /*
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            printf("%d\n", c[i * N + j]);
        }
    }
    */
    
    free(a);
    free(b);
    free(c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    return 0;
}
